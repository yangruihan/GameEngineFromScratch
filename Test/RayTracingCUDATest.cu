
#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result)
                  << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i < max_x) && (j < max_y)) {
        int pixel_index = j * max_x * 3 + i * 3;
        fb[pixel_index + 0] = float(i) / max_x;
        fb[pixel_index + 1] = float(j) / max_y;
        fb[pixel_index + 2] = 0.2f;
    }
}

int main() {
    int image_width = 1024;
    int image_height = 1024;
    int tile_width = 8;
    int tile_height = 8;

    int num_pixels = image_width * image_height;
    size_t fb_size = 3 * num_pixels * sizeof(float);
    float *fb;

    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    dim3 blocks(image_width / tile_width + 1, image_height / tile_height + 1);
    dim3 threads(tile_width, tile_height);
    render<<<blocks, threads>>>(fb, image_width, image_height);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // -------------------------------

    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";

    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * 3 * image_width + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));

    return 0;
}